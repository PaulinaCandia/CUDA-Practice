#include "hip/hip_runtime.h"
%%writefile add_block.cu

#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)// the loop goes through parallel threads
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 256>>>(N, x, y); // the computation will be spread through 256 threads

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}

/*
%%shell

nvcc add_block.cu -o add_block
nvprof ./add_block

Compare the time for the add kernel by looking at the GPU activities field. 
But not surprising since I went from 1 thread to 256 threads. Let’s keep going to get even more performance.
*/